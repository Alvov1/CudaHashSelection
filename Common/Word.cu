#include "Word.h"

namespace HashSelection {
    std::vector<Word> readFileDictionary(const std::filesystem::path& fromLocation) {
        if (!std::filesystem::exists(fromLocation))
            throw std::invalid_argument("Dictionary file is not found");
        if (!std::filesystem::is_regular_file(fromLocation))
            throw std::invalid_argument("Bad dictionary file.");

        return [](const std::filesystem::path& location) {
            std::basic_ifstream<Char> input(location);

            unsigned wordsCount = [&input]() {
                std::basic_string<Char> sizeBuffer(10, Char(0));
                input.getline(sizeBuffer.data(), 10);
                return stoi(sizeBuffer);
            }();

            std::vector<Word> values(wordsCount);
            for (auto& [value, size]: values) {
                static constexpr Char space = [] {
                    if constexpr (std::is_same<Char, char>::value)
                        return ' ';
                    else
                        return L' ';
                }();
                input.getline(value, WordSize, space);

                static constexpr auto strlen = [] {
                    if constexpr (std::is_same<Char, char>::value)
                        return std::strlen;
                    else
                        return std::wcslen;
                }();
                size = strlen(value);
            }

            return values;
        }(fromLocation);
    }
}
