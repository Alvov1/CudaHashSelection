#include "DictionaryOnDevice.h"

DictionaryOnDevice::DictionaryOnDevice(const std::vector<std::string> &words)
        : hostPointersArraySize(words.size()),
          hostPointersArray(std::vector<char*>(words.size() * sizeof(char*))) {
    for(unsigned i = 0; i < hostPointersArraySize; ++i) {
        const auto& current = words[i];
        auto code = hipMalloc(&hostPointersArray[i], current.size() * sizeof(char));
        if(code != hipSuccess)
            throw CudaException(code);
        code = hipMemcpy(hostPointersArray[i], current.c_str(), current.size() * sizeof(char), hipMemcpyHostToDevice);
        if(code != hipSuccess)
            throw CudaException(code);
    }

    size_t arrayMemorySpace = hostPointersArraySize * sizeof(char*);
    auto code = hipMalloc(&devicePointersArray, arrayMemorySpace);
    if(code != hipSuccess)
        throw CudaException(code);
    code = hipMemcpy(devicePointersArray, hostPointersArray.data(), arrayMemorySpace, hipMemcpyHostToDevice);
    if(code != hipSuccess)
        throw CudaException(code);
}

DictionaryOnDevice::~DictionaryOnDevice() {
    for(unsigned i = 0; i < hostPointersArraySize; ++i) {
        auto code = hipFree(hostPointersArray[i]);
        if (code != hipSuccess)
            std::cerr << CudaException(code).what() << std::endl;
    }
    auto code = hipFree(devicePointersArray);
    if(code != hipSuccess)
        std::cerr << CudaException(code).what() << std::endl;
}
