#include "HashSelectionDevice.h"

namespace HashSelection {
    DEVICE bool isVowelDevice(Char sym) {
        if constexpr (std::is_same<Char, char>::value)
        return (sym == 'a' || sym == 'e' || sym == 'i' || sym == 'o' || sym == 'u' || sym == 'y');
        else
        return (sym == L'a' || sym == L'e' || sym == L'i' || sym == L'o' || sym == L'u' || sym == L'y');
    }

    GLOBAL void foundPermutationsDevice(const ExtensionList* words, const unsigned char *withHash, Word* resultPlace) {
        const unsigned threadNumber = threadIdx.x + blockIdx.x * blockDim.x;
        if (threadNumber > 127) return;

        const auto &[pattern, patternSize] = words[threadNumber];

        struct Stack final {
            struct StackElem final {
                Char sym{};
                short amount{};
            } buffer[WordSize];
            uint8_t position{};

            DEVICE uint8_t
            push(Char
                 sym,
                 short amount
            ) {
                if (position + 1 < WordSize)
                    buffer[position] = {sym, amount};
                return ++position;
            }

            DEVICE StackElem

            pop() {
                if (position > 0)
                    return buffer[--position];
                return buffer[0];
            }

            DEVICE bool empty() const { return position == 0; }

            DEVICE Word

            toWord() const {
                Word result{};
                for (uint8_t i = 0; i < position; ++i)
                    result.data[result.size++] = buffer[i].sym;
                return result;
            }
        } stack;
        stack.push(pattern[0], -1);

        while (!stack.empty()) {
            if (stack.position >= patternSize) {
                [&withHash] (const Word& word) {

                } (stack.toWord());
            }
        }
    }

    GLOBAL void foundExtensionsDevice(const Word* words, ExtensionList *extensionsTotal) {
        const unsigned threadNumber = threadIdx.x + blockIdx.x * blockDim.x;
        if (threadNumber > 127) return;

        ExtensionList &currentList = extensionsTotal[threadNumber];

        [&currentList](const Word &forWord) {
            const auto &[pattern, patternSize] = forWord;

            struct Stack final {
                struct StackElem final {
                    Char sym{};
                    uint8_t reps{}, repsNow{};
                } buffer[WordSize]{};
                uint8_t position{};

                DEVICE uint8_t
                push(Char
                sym,
                uint8_t reps, uint8_t
                repsNow) {
                    if (position + 1 < WordSize)
                        buffer[position] = {sym, reps, repsNow};
                    return ++position;
                }
                DEVICE Word

                toWord() const {
                    Word result{};
                    for (uint8_t i = 0; i < position; ++i) {
                        const StackElem &elem = buffer[i];
                        for (uint8_t j = 0; j < elem.repsNow && result.size < WordSize; ++j)
                            result.data[result.size++] = elem.sym;
                    }
                    return result;
                }

                DEVICE StackElem

                pop() {
                    if (position > 0)
                        return buffer[--position];
                    return buffer[0];
                }

                DEVICE bool empty() const { return position == 0; }
            } stack;

            unsigned position = 0;

            for (; !isVowelDevice(pattern[position]) && position < patternSize; ++position)
                stack.push(pattern[position], 1, 1);

            do {
                if (position < patternSize) {
                    /* Count the number of repetition vowels. */
                    uint8_t vowelsCount = 1;
                    for (unsigned i = position + 1;
                         isVowelDevice(pattern[i]) && pattern[i] == pattern[position]; ++vowelsCount, ++i);

                    /* Pushing new value in stack */
                    stack.push(
                            pattern[position],
                            vowelsCount,
                            (isVowelDevice(pattern[position]) && vowelsCount == 1) ? static_cast<uint8_t>(2)
                                                                                   : vowelsCount
                    );
                    position += vowelsCount;
                } else {
                    /* Found new forWord. Pushing into buffer. */
                    currentList.push(stack.toWord());

                    Stack::StackElem current{};
                    do {
                        current = stack.pop();
                        position -= current.reps;
                    } while (!stack.empty() && current.repsNow < 2);

                    if (current.repsNow-- > 1)
                        stack.push(current.sym, current.reps, current.repsNow);
                    position += current.reps;
                }
            } while (!stack.empty());

        }(words[threadNumber]);
    }

    std::optional <Word> runDevice(const std::vector <Word> &words, const Hash::HostSHA256 &hash) {
        const thrust::device_vector<HashSelection::ExtensionList> deviceExtensions = [] (const std::vector<Word>& words) {
            const thrust::device_vector <HashSelection::Word> deviceWords = words;
            thrust::device_vector <HashSelection::ExtensionList> deviceExtensions(words.size());
            Time::cout << "Dicionary loaded onto device and space for extensions is allocated." << Time::endl;

            foundExtensionsDevice<<<8, 16>>>(
                    thrust::raw_pointer_cast(deviceWords.data()),
                    thrust::raw_pointer_cast(deviceExtensions.data()));
            if (hipSuccess != hipDeviceSynchronize())
                throw std::runtime_error("Founding extensions failed.");

            return deviceExtensions;
        } (words);

        Time::cout << "Word extensions found and loaded." << Time::endl;

        const thrust::host_vector<Word> result = [&hash] (const thrust::device_vector<ExtensionList>& deviceExtensions) {
            const thrust::device_vector<unsigned char> deviceHashPattern = [&hash] {
                const auto &data = hash.get();
                return std::vector<unsigned char>(data.begin(), data.end());
            }();
            thrust::device_vector<Word> deviceResult(1);

            foundPermutationsDevice<<<8, 16>>>(
                    thrust::raw_pointer_cast(deviceExtensions.data()),
                    thrust::raw_pointer_cast(deviceHashPattern.data()));
            if (hipSuccess != hipDeviceSynchronize())
                throw std::runtime_error("Founding permutations failed.");

            return deviceResult;
        } (deviceExtensions);

        if(result[0].size > 0) Time::cout << "Completed: " << result[0] << Time::endl;
            else Time::cout << "Failed." << Time::endl;

        return {};
    }
}